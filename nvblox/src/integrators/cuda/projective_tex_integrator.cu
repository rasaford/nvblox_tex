#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/integrators/cuda/projective_integrators_common.cuh"
#include "nvblox/integrators/integrators_common.h"
#include "nvblox/integrators/projective_tex_integrator.h"
#include "nvblox/utils/timing.h"

namespace nvblox {

ProjectiveTexIntegrator::ProjectiveTexIntegrator()
    : ProjectiveIntegratorBase() {
  sphere_tracer_.params().maximum_ray_length_m = max_integration_distance_m_;
  checkCudaErrors(hipStreamCreate(&integration_stream_));
}

ProjectiveTexIntegrator::~ProjectiveTexIntegrator() {
  finish();
  checkCudaErrors(hipStreamDestroy(integration_stream_));
}

void ProjectiveTexIntegrator::finish() const {
  hipStreamSynchronize(integration_stream_);
}

void ProjectiveTexIntegrator::integrateFrame(
    const ColorImage& color_frame, const Transform& T_L_C, const Camera& camera,
    const TsdfLayer& tsdf_layer, TexLayer* tex_layer,
    std::vector<Index3D>* updated_blocks) {
  CHECK_NOTNULL(tex_layer);
  CHECK_EQ(tsdf_layer.block_size(), tex_layer->block_size());

  // Metric truncation distance for this layer
  const float voxel_size =
      tex_layer->block_size() / VoxelBlock<bool>::kVoxelsPerSide;
  const float truncation_distance_m = truncation_distance_vox_ * voxel_size;

  timing::Timer blocks_in_view_timer("tex/integrate/get_blocks_in_view");
  std::vector<Index3D> block_indices =
      getBlocksInView(T_L_C, camera, tex_layer->block_size());
  blocks_in_view_timer.Stop();

  // Check which of these blocks are:
  // - Allocated in the TSDF, and
  // - have at least a single voxel within the truncation band
  // This is because:
  // - We don't allocate new geometry here, we just color existing geometry
  // - We don't color freespace.
  timing::Timer blocks_in_band_timer("tex/integrate/reduce_to_blocks_in_band");
  block_indices = reduceBlocksToThoseInTruncationBand(block_indices, tsdf_layer,
                                                      truncation_distance_m);
  blocks_in_band_timer.Stop();
  
  // Allocate blocks (CPU)
  // We allocate color blocks where
  // - there are allocated TSDF blocks, AND
  // - these blocks are within the truncation band
  timing::Timer allocate_blocks_timer("tex/integrate/allocate_blocks");
  allocateBlocksWhereRequired(block_indices, tex_layer);
  allocate_blocks_timer.Stop();

  timing::Timer update_neighbor_block_indices_timer(
      "tex/integrate/neighbor_block_indices");
  updateNeighborIndicies(tsdf_layer, block_indices);
  update_neighbor_block_indices_timer.Stop();

  tex_layer->waitForPrefetch();
  // Update normal directions for all voxels which do not have a voxel dir set
  // already
  timing::Timer update_normals_timer("tex/integrate/update_normals");
  updateVoxelNormalDirections(tsdf_layer, tex_layer, block_indices,
                              truncation_distance_m);
  update_normals_timer.Stop();

  // Create a synthetic depth image
  timing::Timer sphere_trace_timer("tex/integrate/sphere_trace");
  std::shared_ptr<const DepthImage> synthetic_depth_image_ptr =
      sphere_tracer_.renderImageOnGPU(
          camera, T_L_C, tsdf_layer, truncation_distance_m, MemoryType::kDevice,
          depth_render_ray_subsampling_factor_);
  sphere_trace_timer.Stop();

  // Update identified blocks
  // Calls out to the child-class implementing the integation (GPU)
  timing::Timer update_blocks_timer("tex/integrate/update_blocks");
  updateBlocks(block_indices, color_frame, *synthetic_depth_image_ptr, T_L_C,
               camera, truncation_distance_m, tsdf_layer, tex_layer);
  update_blocks_timer.Stop();

  if (updated_blocks != nullptr) {
    *updated_blocks = block_indices;
  }
}

void ProjectiveTexIntegrator::updateNeighborIndicies(
    const TsdfLayer& tsdf_layer, const std::vector<Index3D>& block_indices) {
  const int new_size = block_indices.size() * tex::neighbor::kCubeNeighbors;
  tsdf_block_ptrs_host_.resize(new_size);
  tsdf_block_ptrs_device_.resize(new_size);

  for (int i = 0; i < block_indices.size(); ++i) {
    for (int j = 0; j < tex::neighbor::kCubeNeighbors; ++j) {
      Index3D offset = tex::neighbor::blockOffsetFromNeighborIndex(j);
      tsdf_block_ptrs_host_[i * tex::neighbor::kCubeNeighbors + j] =
          tsdf_layer.getBlockAtIndex(block_indices[i] + offset).get();
    }
  }
  tsdf_block_ptrs_device_ = tsdf_block_ptrs_host_;
}

__device__ float computeMeasurementWeight(const TexVoxel* tex_voxel,
                                          const Transform& T_C_L,
                                          const Vector3f& voxel_center,
                                          const Vector2f& u_px,
                                          const float u_px_depth) {
  // Area based weighting
  // Minimum depth of scanning (m). I.e. closest we will get to a point.
  constexpr float MIN_DEPTH = .1f;
  // Smoothing for the deviation in normal direction we accept for w_area
  constexpr float SIMGA_AREA = 2.f;
  // Smoothing for the deviation in normal direction we accept for w_angle
  constexpr float SIMGA_ANGLE = 1.f;
  constexpr float MIN_W_AREA = .1f;   // GAMAM_AREA in TextureFusion Paper
  constexpr float MIN_W_ANGLE = .1f;  // GAMAM_ANGLE in TextureFusion Paper

  Vector3f view_dir = (T_C_L.translation() - voxel_center).normalized();
  float normal_align =
      tex::texDirToWorldVector(tex_voxel->dir).dot(view_dir);  // in [-1, 1]
  float depth_clipped = fmax(u_px_depth, MIN_DEPTH);
  // rho is the product of the alignment of the view direction with the surface
  // normal at the given voxel and the clipped inverse depth. I.e. voxels that
  // we look at head on and are close to the camera are preferred
  float rho =
      powf(MIN_DEPTH / depth_clipped, 2.f) * normal_align;  // in [-1, 1]

  // w_area is a bell curve centered at 1 with rho as a parameter. So the closer
  // rho is to one, the more weight we assign it. SIGMA_AREA controlls the
  // sharpness of the falloff at around the mean 1.
  // clang-format off
  float w_area = fmax(
      expf(-powf((1 - rho) / SIMGA_AREA, 2.f)), 
      MIN_W_AREA
  ); // in [MIN_W_AREA, 1]
  // clang-format on

  // View angle based weighting
  // clang-format off
  float w_angle = fmax(
    expf(-powf((1 - normal_align) / SIMGA_ANGLE, 2.f)),
    MIN_W_ANGLE
  ); // in [MIN_W_ANGLE, 1]
  // clang-format on

  return w_area * w_angle;  // in [0, 1]
}

__device__ inline void updateTexel(const Color& color_measured,
                                   TexVoxel* tex_voxel,
                                   const Index2D& texel_idx,
                                   const float measurement_weight,
                                   const float max_weight) {
  tex_voxel->color(texel_idx) =
      blendTwoColors(tex_voxel->color(texel_idx), tex_voxel->weight,
                     color_measured, measurement_weight);
}

__device__ const TsdfVoxel* getNeighborVoxelAtIndex(
    const TsdfBlock** blocks, const Index3D& voxel_index) {
  const int block_idx = blockIdx.x;

  // create a local copy of the voxel index, since we are going to be modifying
  // it.
  Index3D voxel_idx = voxel_index;
  Index3D block_offset{0, 0, 0};
  constexpr int voxels_per_side = static_cast<int>(TsdfBlock::kVoxelsPerSide);
  for (int i = 0; i < 3; ++i) {
    if (voxel_idx[i] >= voxels_per_side) {
      voxel_idx[i] -= voxels_per_side;
      block_offset[i] = 1;
    } else if (voxel_idx[i] < 0) {
      voxel_idx[i] += voxels_per_side;
      block_offset[i] = -1;
    }
  }
  // get the neighboring voxel either from a neighboring block if it's outside
  // the current one, or get it directly from the current block
  int linear_neighbor_idx =
      tex::neighbor::neighborBlockIndexFromOffset(block_offset);
  const TsdfBlock* block =
      blocks[block_idx * tex::neighbor::kCubeNeighbors + linear_neighbor_idx];

  if (block == nullptr) {
    return nullptr;
  }
  return &block->voxels[voxel_idx.x()][voxel_idx.y()][voxel_idx.z()];
}

/**
 * @brief Gets TSDF values at two positions at the same time. Where the surface
 * is sampled around position1 and extrapolated to position2.
 * This saves time, compared to two separate interpolations
 *
 * @param blocks neihbor blocks
 * @param position1 first position to interpolate the surface distance at
 * @param position2 second position to interpolate the surface distance at
 * @param index1 voxel index of position1
 * @param voxel_size
 * @param sdf1 output sdf at position1
 * @param sdf2 output sdf at position2
 * @return __device__ if the interpolation was successful
 */
__device__ bool getTSDFValues(const TsdfBlock** blocks,
                              const Vector3f& position1,
                              const Vector3f& position2, const Index3D& index1,
                              const float voxel_size, float* sdf1,
                              float* sdf2) {
  Vector3f normalized_pos1 = position1 / voxel_size;
  Vector3f normalized_pos2 = position2 / voxel_size;
  // SDF interpolation Weights are in range [0,1] for each axis
  Vector3f weight1 = normalized_pos1.array() - normalized_pos1.array().floor();
  Vector3f weight2 = normalized_pos2.array() - normalized_pos1.array().floor();
  // clear sdf values
  (*sdf1) = 0.f;
  (*sdf2) = 0.f;

  const TsdfVoxel* v;
  // clang-format off
  v = getNeighborVoxelAtIndex(blocks, index1);                    if (v == nullptr) return false;     (*sdf1) += (1.0f - weight1.x()) * (1.0f - weight1.y()) * (1.0f - weight1.z()) * v->distance; 
                                                                                                      (*sdf2) += (1.0f - weight2.x()) * (1.0f - weight2.y()) * (1.0f - weight2.z()) * v->distance; 
  v = getNeighborVoxelAtIndex(blocks, index1 + Index3D(1, 0, 0)); if (v == nullptr) return false;     (*sdf1) += weight1.x()          * (1.0f - weight1.y()) * (1.0f - weight1.z()) * v->distance; 
                                                                                                      (*sdf2) += weight2.x()          * (1.0f - weight2.y()) * (1.0f - weight2.z()) * v->distance; 
  v = getNeighborVoxelAtIndex(blocks, index1 + Index3D(0, 1, 0)); if (v == nullptr) return false;     (*sdf1) += (1.0f - weight1.x()) * weight1.y()          * (1.0f - weight1.z()) * v->distance; 
                                                                                                      (*sdf2) += (1.0f - weight2.x()) * weight2.y()          * (1.0f - weight2.z()) * v->distance; 
  v = getNeighborVoxelAtIndex(blocks, index1 + Index3D(0, 0, 1)); if (v == nullptr) return false;     (*sdf1) += (1.0f - weight1.x()) * (1.0f - weight1.y()) * weight1.z()          * v->distance; 
                                                                                                      (*sdf2) += (1.0f - weight2.x()) * (1.0f - weight2.y()) * weight2.z()          * v->distance; 
  v = getNeighborVoxelAtIndex(blocks, index1 + Index3D(1, 1, 0)); if (v == nullptr) return false;     (*sdf1) += weight1.x()          * weight1.y()          * (1.0f - weight1.z()) * v->distance; 
                                                                                                      (*sdf2) += weight2.x()          * weight2.y()          * (1.0f - weight2.z()) * v->distance; 
  v = getNeighborVoxelAtIndex(blocks, index1 + Index3D(0, 1, 1)); if (v == nullptr) return false;     (*sdf1) += (1.0f - weight1.x()) * weight1.y()          * weight1.z()          * v->distance; 
                                                                                                      (*sdf2) += (1.0f - weight2.x()) * weight2.y()          * weight2.z()          * v->distance; 
  v = getNeighborVoxelAtIndex(blocks, index1 + Index3D(1, 0, 1)); if (v == nullptr) return false;     (*sdf1) += weight1.x()          * (1.0f - weight1.y()) * weight1.z()          * v->distance; 
                                                                                                      (*sdf2) += weight2.x()          * (1.0f - weight2.y()) * weight2.z()          * v->distance; 
  v = getNeighborVoxelAtIndex(blocks, index1 + Index3D(1, 1, 1)); if (v == nullptr) return false;     (*sdf1) += weight1.x()          * weight1.y()          * weight1.z()          * v->distance; 
                                                                                                      (*sdf2) += weight2.x()          * weight2.y()          * weight2.z()          * v->distance;
  // clang-format on
  return true;
}

/**
 * @brief For the line between (x1, y1), (x2, y2) we find the intersection with
 * the x-axis.
 * Solves y - y1 = (y2 - y1) / (x2 - x1) * (x - x1) = 0 for x
 *
 * @param x1 x coordinate of point1
 * @param x2 x coordinate of point2
 * @param y1 y coordinate of point1
 * @param y2 y coordinate of point2
 * @return __device__
 */
__device__ inline float findIntersectionLinear(const float x1, const float x2,
                                               const float y1, const float y2) {
  return x1 + (y1 / (y1 - y2)) * (x2 - x1);
}

/**
 * @brief Finds the intersection of the ray starting at the given postion along
 * the given direction with the TSDF surface. The distance along this ray is
 * returned.
 *
 * @param blocks neighbor blocks
 * @param voxel_size voxel size
 * @param position stating position
 * @param direction ray directions
 * @param distance dinstance from position along dir until the intersection
 * @return __device__ if an intersection could be found along the ray
 */
__device__ bool raycastToSurface(const TsdfBlock** blocks,
                                 const float voxel_size,
                                 const Vector3f& position,
                                 const TexVoxel::Dir& direction,
                                 float* distance) {
  const Index3D voxel_idx = Index3D(threadIdx.z, threadIdx.y, threadIdx.x);
  Vector3f next_voxel = tex::texDirToWorldVector(direction) * voxel_size;
  float sdf1, sdf2;

  if (getTSDFValues(blocks, position, position + next_voxel, voxel_idx,
                    voxel_size, &sdf1, &sdf2)) {
    (*distance) = findIntersectionLinear(0.f, voxel_size, sdf1, sdf2);
    return true;
  }
  return false;
}

__global__ void integrateBlocks(
    const Index3D* block_indices_device_ptr, const Camera camera,
    const Color* color_image, const int color_rows, const int color_cols,
    const float* depth_image, const int depth_rows, const int depth_cols,
    const Transform T_C_L, const float block_size,
    const float truncation_distance_m, const float max_weight,
    const float max_integration_distance, const int depth_subsample_factor,
    const TsdfBlock** tsdf_blocks, TexBlock** tex_ptrs) {
  // Get - the image-space projection of the voxel center associated with this
  // thread
  Eigen::Vector2f u_px;
  float voxel_depth_m;
  if (!projectThreadVoxel(block_indices_device_ptr, camera, T_C_L, block_size,
                          &u_px, &voxel_depth_m)) {
    return;
  }

  // If voxel further away than the limit, skip this voxel
  if (max_integration_distance > 0.0f) {
    if (voxel_depth_m > max_integration_distance) {
      return;
    }
  }

  // Get - the depth of the voxel center
  //     - Also check if the voxel projects inside the image
  const Vector2f u_px_depth = u_px / static_cast<float>(depth_subsample_factor);
  float surface_depth_m;
  if (!interpolation::interpolate2DLinear<float>(
          depth_image, u_px_depth, depth_rows, depth_cols, &surface_depth_m)) {
    return;
  }

  // Occlusion testing
  // Get the distance of the voxel from the rendered surface. If outside
  // truncation band, skip.
  const float voxel_distance_from_surface = surface_depth_m - voxel_depth_m;
  if (fabsf(voxel_distance_from_surface) > truncation_distance_m) {
    return;
  }

  // Get the Voxel we'll update in this thread
  // NOTE(alexmillane): Note that we've reverse the voxel indexing order such
  // that adjacent threads (x-major) access adjacent memory locations in the
  // block (z-major).
  TexVoxel* voxel_ptr =
      &(tex_ptrs[blockIdx.x]->voxels[threadIdx.z][threadIdx.y][threadIdx.x]);

  // NOTE(rasaford): If the current voxel has not been assigned a texture plane
  // direction, it must not be on the truncation band --> skip it
  if (!voxel_ptr->isInitialized()) {
    return;
  }

  // Update the weight of each tex voxel once per voxel (instead of once per
  // texel) as the average of the new and old weights
  const Index3D& block_idx = block_indices_device_ptr[blockIdx.x];
  const Index3D voxel_idx = Index3D(threadIdx.z, threadIdx.y, threadIdx.x);
  const Vector3f voxel_center = getCenterPostionFromBlockIndexAndVoxelIndex(
      block_size, block_idx, voxel_idx);

  // float measurement_weight = 1.f;

  float measurement_weight = computeMeasurementWeight(
      voxel_ptr, T_C_L, voxel_center, u_px, surface_depth_m);
  Color image_value = Color::Black();
  Index2D texel_idx{0, 0};
  Vector3f texel_pos = Vector3f::Zero();
  Vector3f surface_pos = Vector3f::Zero();
  for (int row = 0; row < voxel_ptr->kPatchWidth; ++row) {
    for (int col = 0; col < voxel_ptr->kPatchWidth; ++col) {
      texel_idx = Index2D(row, col);
      image_value = Color::Black();

      // Orthogonal projection of TexVoxel tile to SDF surface
      texel_pos = getCenterPositionForTexel(block_size, block_idx, voxel_idx,
                                            texel_idx, voxel_ptr->dir);

      float distance;
      if (!raycastToSurface(tsdf_blocks, block_size / TsdfBlock::kVoxelsPerSide,
                            texel_pos, voxel_ptr->dir, &distance)) {
        continue;
      }

      surface_pos =
          texel_pos + distance * tex::texDirToWorldVector(voxel_ptr->dir);

      // Project the current texel_idx to image space. If it's outside the
      // image, go to the next texel.
      surface_pos = T_C_L * surface_pos;
      if (!camera.project(surface_pos, &u_px)) {
        continue;
      }
      // sample the color at the interpolated point
      if (!interpolation::interpolate2DLinear<Color>(
              color_image, u_px, color_rows, color_cols, &image_value)) {
        continue;
      }
      // update the texel color
      updateTexel(image_value, voxel_ptr, texel_idx, measurement_weight,
                  max_weight);
    }
  }
  // Since the voxel_weight is read when updating the texels, it must be updated
  // after all texels. This is a non-saturating filter version of the weighting
  // rule described in TextureFusion
  voxel_ptr->weight = (measurement_weight + voxel_ptr->weight) / 2;
}

void ProjectiveTexIntegrator::updateBlocks(
    const std::vector<Index3D>& block_indices, const ColorImage& color_frame,
    const DepthImage& depth_frame, const Transform& T_L_C, const Camera& camera,
    const float truncation_distance_m, const TsdfLayer& tsdf_layer,
    TexLayer* tex_layer_ptr) {
  CHECK_NOTNULL(tex_layer_ptr);
  CHECK_EQ(color_frame.rows() % depth_frame.rows(), 0);
  CHECK_EQ(color_frame.cols() % depth_frame.cols(), 0);

  if (block_indices.empty()) {
    return;
  }
  const int num_blocks = block_indices.size();
  const int depth_subsampling_factor = color_frame.rows() / depth_frame.rows();
  CHECK_EQ(color_frame.cols() / depth_frame.cols(), depth_subsampling_factor);

  // Expand the buffers when needed
  if (num_blocks > block_indices_device_.size()) {
    const int new_size = static_cast<int>(kBufferExpansionFactor * num_blocks);
    block_indices_device_.reserve(new_size);
    tex_block_ptrs_device_.reserve(new_size);
    block_indices_host_.reserve(new_size);
    tex_block_ptrs_host_.reserve(new_size);
  }

  // Stage on the host pinned memory
  block_indices_host_ = block_indices;
  tex_block_ptrs_host_ = getBlockPtrsFromIndices(block_indices, tex_layer_ptr);

  // Transfer to the device
  block_indices_device_ = block_indices_host_;
  tex_block_ptrs_device_ = tex_block_ptrs_host_;

  // We need the inverse transform in the kernel
  const Transform T_C_L = T_L_C.inverse();

  // Kernel call - One ThreadBlock launched per VoxelBlock
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const dim3 kThreadsPerBlock(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  const int num_thread_blocks = block_indices.size();
  // clang-format off
  integrateBlocks<<<num_thread_blocks, kThreadsPerBlock, 0, integration_stream_>>>(
      block_indices_device_.data(),
      camera,
      color_frame.dataConstPtr(),
      color_frame.rows(),
      color_frame.cols(),
      depth_frame.dataConstPtr(),
      depth_frame.rows(),
      depth_frame.cols(),
      T_C_L,
      tex_layer_ptr->block_size(),
      truncation_distance_m,
      max_weight_,
      max_integration_distance_m_,
      depth_subsampling_factor,
      tsdf_block_ptrs_device_.data(),
      tex_block_ptrs_device_.data());
  // clang-format on
  checkCudaErrors(hipStreamSynchronize(integration_stream_));
  checkCudaErrors(hipPeekAtLastError());

}

std::vector<Index3D>
ProjectiveTexIntegrator::reduceBlocksToThoseInTruncationBand(
    const std::vector<Index3D>& block_indices, const TsdfLayer& tsdf_layer,
    const float truncation_distance_m) {
  // Check 1) Are the blocks allocated
  // - performed on the CPU because the hash-map is on the CPU
  std::vector<Index3D> block_indices_check_1;
  block_indices_check_1.reserve(block_indices.size());
  for (const Index3D& block_idx : block_indices) {
    if (tsdf_layer.isBlockAllocated(block_idx)) {
      block_indices_check_1.push_back(block_idx);
    }
  }

  if (block_indices_check_1.empty()) {
    return block_indices_check_1;
  }

  // Check 2) Does each of the blocks have a voxel within the truncation band
  // - performed on the GPU because the blocks are there
  // Get the blocks we need to check
  std::vector<const TsdfBlock*> block_ptrs =
      getBlockPtrsFromIndices(block_indices_check_1, tsdf_layer);

  const int num_blocks = block_ptrs.size();

  // Expand the buffers when needed
  if (num_blocks > truncation_band_block_ptrs_device_.size()) {
    const int new_size = static_cast<int>(kBufferExpansionFactor * num_blocks);
    truncation_band_block_ptrs_host_.reserve(new_size);
    truncation_band_block_ptrs_device_.reserve(new_size);
    block_in_truncation_band_device_.reserve(new_size);
    block_in_truncation_band_host_.reserve(new_size);
  }

  // Host -> Device
  truncation_band_block_ptrs_host_ = block_ptrs;
  truncation_band_block_ptrs_device_ = truncation_band_block_ptrs_host_;

  // Prepare output space
  block_in_truncation_band_device_.resize(num_blocks);

  // Do the check on GPU
  // Kernel call - One ThreadBlock launched per VoxelBlock
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const dim3 kThreadsPerBlock(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  const int num_thread_blocks = num_blocks;
  // clang-format off
  checkBlocksInTruncationBand<<<num_thread_blocks, kThreadsPerBlock, 0, integration_stream_>>>(
      truncation_band_block_ptrs_device_.data(),
      truncation_distance_m,
      block_in_truncation_band_device_.data());
  // clang-format on
  checkCudaErrors(hipStreamSynchronize(integration_stream_));
  checkCudaErrors(hipPeekAtLastError());

  // Copy results back
  block_in_truncation_band_host_ = block_in_truncation_band_device_;

  // Filter the indices using the result
  std::vector<Index3D> block_indices_check_2;
  block_indices_check_2.reserve(block_indices_check_1.size());
  for (int i = 0; i < block_indices_check_1.size(); i++) {
    if (block_in_truncation_band_host_[i]) {
      block_indices_check_2.push_back(block_indices_check_1[i]);
    }
  }

  return block_indices_check_2;
}

void ProjectiveTexIntegrator::updateVoxelNormalDirections(
    const TsdfLayer& tsdf_layer, TexLayer* tex_layer_ptr,
    const std::vector<Index3D>& block_indices,
    const float truncation_distance_m) {
  if (block_indices.empty()) {
    return;
  }
  // Get the pointers for the indexed blocks from both
  // - The tsdf layer: Since all Voxels are already integrated here, we read
  // from this layer to estimate the normal direcitonA
  // - The TexBlock layer: We write the updated directions to this layer for
  // all new blocks
  // NOTE(rasaford) Even though we do not modify TsdfLayer, we have to drop
  // const here due to the way the GPUHashMap works
  // TsdfLayer* tsdf_layer_non_const = const_cast<TsdfLayer*>(&tsdf_layer);
  // std::vector<TsdfBlock*> tsdf_block_ptrs =
  //     getBlockPtrsFromIndices(block_indices, tsdf_layer_non_const);
  std::vector<TexBlock*> tex_block_ptrs =
      getBlockPtrsFromIndices(block_indices, tex_layer_ptr);

  // // We assume that a TsdfBlock at index i corresponds to a TexBlock at i.
  // This
  // // cannot be the case if the two vectors don't have the same number of
  // // elements
  // CHECK_EQ(tsdf_block_ptrs.size(), tex_block_ptrs.size());

  const int num_blocks = block_indices.size();
  // Expand the buffers when needed
  if (num_blocks > update_normals_tex_block_prts_device_.size()) {
    const int new_size = static_cast<int>(kBufferExpansionFactor * num_blocks);
    update_normals_tex_block_prts_device_.reserve(new_size);
    update_normals_tex_block_prts_host_.reserve(new_size);
    update_normals_block_indices_device_.reserve(new_size);
    update_normals_block_indices_host_.reserve(new_size);
  }

  // Host -> Device
  update_normals_tex_block_prts_host_ = tex_block_ptrs;
  update_normals_tex_block_prts_device_ = update_normals_tex_block_prts_host_;
  update_normals_block_indices_host_ = block_indices;
  update_normals_block_indices_device_ = update_normals_block_indices_host_;

  tex::updateTexVoxelDirectionsGPU(
      tsdf_block_ptrs_device_, update_normals_block_indices_device_,
      update_normals_tex_block_prts_device_, num_blocks, integration_stream_,
      tsdf_layer.block_size(), tsdf_layer.voxel_size());
}

}  // namespace nvblox
