#include "hip/hip_runtime.h"
#include <vector>

// #include <thrust/device_vector.h>
// #include <thrust/host_vector.h>
#include "nvblox/core/common_names.h"
#include "nvblox/core/layer.h"
#include "nvblox/gpu_hash/cuda/gpu_hash_interface.cuh"
#include "nvblox/gpu_hash/cuda/gpu_indexing.cuh"
#include "nvblox/ray_tracing/sphere_tracer.h"
#include "nvblox/tex/tex_integrator_kernels.h"

namespace nvblox {
namespace tex {

__device__ const TsdfVoxel* getVoxel(const TsdfBlock** blocks,
                                     const Index3D& voxel_index) {
  const int block_idx = blockIdx.x;

  // create a local copy of the voxel index, since we are going to be modifying
  // it.
  Index3D voxel_idx = voxel_index;
  Index3D block_offset{0, 0, 0};
  constexpr int voxels_per_side = static_cast<int>(TsdfBlock::kVoxelsPerSide);
  for (int i = 0; i < 3; ++i) {
    if (voxel_idx[i] >= voxels_per_side) {
      voxel_idx[i] -= voxels_per_side;
      block_offset[i] = 1;
    } else if (voxel_idx[i] < 0) {
      voxel_idx[i] += voxels_per_side;
      block_offset[i] = -1;
    }
  }

  // We cannot look more than one block in any direction
  if ((block_offset.array() > 1).any() || (block_offset.array() < -1).any() ||
      (voxel_idx.array() < 0).any() ||
      (voxel_idx.array() >= TsdfBlock::kVoxelsPerSide).any()) {
    printf(
        "voxel_index (%d, %d, %d), voxel_idx (%d, %d, %d), block_offset(%d, "
        "%d, %d)\n",
        voxel_index[0], voxel_index[1], voxel_index[2], voxel_idx[0],
        voxel_idx[1], voxel_idx[2], block_offset[0], block_offset[1],
        block_offset[2]);
    return nullptr;
  }
  // get the neighboring voxel either from a neighboring block if it's outside
  // the current one, or get it directly from the current block
  int linear_neighbor_idx =
      neighbor::neighborBlockIndexFromOffset(block_offset);
  const TsdfBlock* block =
      blocks[block_idx * neighbor::kCubeNeighbors + linear_neighbor_idx];

  if (block == nullptr) {
    return nullptr;
  }
  return &block->voxels[voxel_idx.x()][voxel_idx.y()][voxel_idx.z()];
}

/**
 * @brief
 *
 * @param blocks
 * @param position has to be inside the givne by voxel_idx
 * @param voxel_idx
 * @param dist
 * @return __device__
 */
__device__ bool trilinearInterpolation(const TsdfBlock** blocks,
                                       const Vector3f& position,
                                       const Index3D voxel_idx,
                                       const float voxel_size, float* dist) {
  const Vector3f normalized_pos = position / voxel_size;
  const Vector3f weight =
      normalized_pos.array() - normalized_pos.array().floor();

  (*dist) = 0.f;
  const TsdfVoxel* v;
  // clang-format off
  v = getVoxel(blocks, voxel_idx + Index3D(0, 0, 0));   if (v == nullptr || v->weight == 0.f) return false;   (*dist) += (1.f - weight.x()) * (1.f - weight.y()) * (1.f - weight.z()) * v->distance;
  v = getVoxel(blocks, voxel_idx + Index3D(1, 0, 0));   if (v == nullptr || v->weight == 0.f) return false;   (*dist) += weight.x()         * (1.f - weight.y()) * (1.f - weight.z()) * v->distance;
  v = getVoxel(blocks, voxel_idx + Index3D(0, 1, 0));   if (v == nullptr || v->weight == 0.f) return false;   (*dist) += (1.f - weight.x()) * weight.y()         * (1.f - weight.z()) * v->distance;
  v = getVoxel(blocks, voxel_idx + Index3D(0, 0, 1));   if (v == nullptr || v->weight == 0.f) return false;   (*dist) += (1.f - weight.x()) * (1.f - weight.y()) *  weight.z()        * v->distance;
  v = getVoxel(blocks, voxel_idx + Index3D(1, 1, 0));   if (v == nullptr || v->weight == 0.f) return false;   (*dist) += weight.x()         * weight.y()         * (1.f - weight.z()) * v->distance;
  v = getVoxel(blocks, voxel_idx + Index3D(0, 1, 1));   if (v == nullptr || v->weight == 0.f) return false;   (*dist) += (1.f - weight.x()) * weight.y()         *  weight.z()        * v->distance;
  v = getVoxel(blocks, voxel_idx + Index3D(1, 0, 1));   if (v == nullptr || v->weight == 0.f) return false;   (*dist) += weight.x()         * (1.f - weight.y()) *  weight.z()        * v->distance;
  v = getVoxel(blocks, voxel_idx + Index3D(1, 1, 1));   if (v == nullptr || v->weight == 0.f) return false;   (*dist) += weight.x()         *  weight.y()        *  weight.z()        * v->distance;
  // clang-format on

  return true;
}

/**
 * @brief Computes TSDF gradient for the current position.
 * If a vertex does not have 6 neighbors, the gradient will be (0, 0, 0).
 *
 * @param gpu_hash_index_to_ptr
 * @param position
 * @param block_size
 * @param voxel_size
 * @param gradient gradient at the given position
 * @return if the computed gradient is valid (i.e. the voxel at this position
 * has at least 6 neighbors)
 */
__device__ bool computeTSDFGradient(const TsdfBlock** neighbor_blocks,
                                    const Index3D& voxel_idx,
                                    const Vector3f& position,
                                    const float voxel_size,
                                    Vector3f& gradient) {
  // voxel size is block size divided by number of blocks per side
  // const float voxel_size =
  //     block_size / static_cast<float>(tsdf_block->kVoxelsPerSide);
  const float voxel_size_half = 0.5f * voxel_size;
  const float v_quarter = 0.25f * voxel_size;
  float dist_x_plus = 0, dist_x_minus = 0, dist_y_plus = 0, dist_y_minus = 0,
        dist_z_plus = 0, dist_z_minus = 0;
  bool valid = true;

  const TsdfVoxel* v;
  // get tsdf values for each neighboring voxel to the current one
  // clang-format off
  valid &= trilinearInterpolation(neighbor_blocks, position + Vector3f(v_quarter, 0.f,        0.f), voxel_idx, voxel_size, &dist_x_plus);
  valid &= trilinearInterpolation(neighbor_blocks, position + Vector3f(0.f,       v_quarter,  0.f), voxel_idx, voxel_size, &dist_y_plus);
  valid &= trilinearInterpolation(neighbor_blocks, position + Vector3f(0.f,       0.f,  v_quarter), voxel_idx, voxel_size, &dist_z_plus);
  valid &= trilinearInterpolation(neighbor_blocks, position - Vector3f(v_quarter, 0.f,        0.f), voxel_idx, voxel_size, &dist_x_minus);
  valid &= trilinearInterpolation(neighbor_blocks, position - Vector3f(0.f,       v_quarter,  0.f), voxel_idx, voxel_size, &dist_y_minus);
  valid &= trilinearInterpolation(neighbor_blocks, position - Vector3f(0.f,       0.f,  v_quarter), voxel_idx, voxel_size, &dist_z_minus);
  // clang-format on

  if (!valid) {
    return false;
  }

  // approximate gradient by finite differences
  gradient << (dist_x_plus - dist_x_minus) / voxel_size_half,
      (dist_y_plus - dist_y_minus) / voxel_size_half,
      (dist_z_plus - dist_z_minus) / voxel_size_half;
  return true;
}

/**
 * @brief quantizes the given direction vector (normalized) into one of
 * TexVoxel::Dir directions
 *
 * @param normal **normalized** direction vector
 * @return quantized direction
 */
__device__ inline TexVoxel::Dir quantizeDirection(const Vector3f& dir) {
  const Vector3f abs_dir = dir.cwiseAbs();
  TexVoxel::Dir res;
  if (abs_dir(0) >= abs_dir(1) && abs_dir(0) >= abs_dir(2)) {
    res = dir(0) < 0 ? TexVoxel::Dir::X_MINUS : res = TexVoxel::Dir::X_PLUS;
  } else if (abs_dir(1) >= abs_dir(2)) {
    res = dir(1) < 0 ? TexVoxel::Dir::Y_MINUS : TexVoxel::Dir::Y_PLUS;
  } else {
    res = dir(2) < 0 ? TexVoxel::Dir::Z_MINUS : TexVoxel::Dir::Z_PLUS;
  }
  return res;
}

/**
 * @brief Computes the weight (confidence) we have in the quantized direction
 * given the true gradient direction
 *
 * @param dir quantized direction of the given gradient
 * @param gradient **normalized** true gradient direction
 * @return __device__ weight \in [-1, 1], if dir is the most likely one: weight
 * \in [0, 1]
 */
__device__ inline float computeDirWeight(const TexVoxel::Dir dir,
                                         const Vector3f& gradient) {
  return texDirToWorldVector(dir).dot(gradient);
}

/**
 * @brief Updates the direction values for all TexVoxels where this has not
 * been set yet
 *
 * @param tsdf_block_ptrs
 * @param tex_block_ptrs
 * @param block_size
 */
__global__ void setTexVoxelDirsfromTsdfGradient(
    const TsdfBlock** neighbor_blocks, TexBlock** tex_block_ptrs,
    const Index3D* block_indices, const float block_size,
    const float voxel_size) {
  // Get the Voxels we'll check in this thread
  TexBlock* tex_block = tex_block_ptrs[blockIdx.x];
  Index3D block_idx = block_indices[blockIdx.x];
  Index3D voxel_idx = Index3D(threadIdx.z, threadIdx.y, threadIdx.x);
  TexVoxel* tex_voxel =
      &(tex_block->voxels[voxel_idx[0]][voxel_idx[1]][voxel_idx[2]]);

  // only update the direction for voxels where we are not yet very confident in
  // thier direction
  // if (tex_voxel->dir_weight >= TexVoxel::DIR_THRESHOLD) return;

  Vector3f position = getCenterPostionFromBlockIndexAndVoxelIndex(
      block_size, block_idx, voxel_idx);

  // Since we are working in an TSDF, where the distance of each voxel to the
  // surface implicitly defines the surface boundary, the normal of each voxel
  // is just the normalized gradient.
  Vector3f gradient;
  const bool valid_gradient = computeTSDFGradient(
      neighbor_blocks, voxel_idx, position, voxel_size, gradient);
  const double gradient_norm = gradient.norm();
  if (!valid_gradient || gradient_norm <= 0) return;

  gradient /= gradient_norm;  // normalize gradient inplace

  // Since the quanization of the normalized gradient to the 6 differnt major
  // axis directions in TexVoxel::Dir always introduces an error, we track the
  // confidence in the computed quantization. If we are sufficiently more
  // confident in a newly computed gradient direction we upate the associated
  // texture
  TexVoxel::Dir dir = quantizeDirection(gradient);
  float dir_weight = computeDirWeight(dir, gradient);
  if (TexVoxel::DIR_THRESHOLD * dir_weight >= tex_voxel->dir_weight) {
    // printf("new_dir_weight: %f, tex_voxel->dir_weight: %f\n", dir_weight,
    // tex_voxel->dir_weight);
    tex_voxel->updateDir(dir, dir_weight);
  }
  // printf("gradient: (%f %f %f), dir: %d\n", gradient[0], gradient[1],
  // gradient[2], tex_voxel.dir);
}

/**
 * @brief Computes a historgram where the index is the diretion of each
 * neighboring voxel and the value is the accumulated confidence for all voxels
 * with that direction
 *
 * @param gpu_hash_index_to_ptr
 * @param position
 * @param block_size
 * @param voxel_size
 * @param histogram
 * @return __device__ if the computed histogram is valid, i.e. there exist
 * neighboring voxels in all directions of the given position
 */
__device__ bool computeDirHistogram(
    const Index3DDeviceHashMapType<TexBlock>& gpu_hash_index_to_ptr,
    const Vector3f& position, const float block_size, const float voxel_size,
    Vector7f& weights, Vector7i& frequencies) {
  bool valid = true;

  // clang-format off
  TexVoxel *voxel;
  if (valid &= getVoxelAtPosition<TexVoxel>(gpu_hash_index_to_ptr, position                                 , block_size, &voxel)) { const auto idx = static_cast<int>(voxel->dir); frequencies[idx]++; weights[idx] += voxel->dir_weight; }
  if (valid &= getVoxelAtPosition<TexVoxel>(gpu_hash_index_to_ptr, position + Vector3f(voxel_size, 0.f, 0.f), block_size, &voxel)) { const auto idx = static_cast<int>(voxel->dir); frequencies[idx]++; weights[idx] += voxel->dir_weight; }
  if (valid &= getVoxelAtPosition<TexVoxel>(gpu_hash_index_to_ptr, position + Vector3f(0.f, voxel_size, 0.f), block_size, &voxel)) { const auto idx = static_cast<int>(voxel->dir); frequencies[idx]++; weights[idx] += voxel->dir_weight; }
  if (valid &= getVoxelAtPosition<TexVoxel>(gpu_hash_index_to_ptr, position + Vector3f(0.f, 0.f, voxel_size), block_size, &voxel)) { const auto idx = static_cast<int>(voxel->dir); frequencies[idx]++; weights[idx] += voxel->dir_weight; }
  if (valid &= getVoxelAtPosition<TexVoxel>(gpu_hash_index_to_ptr, position - Vector3f(voxel_size, 0.f, 0.f), block_size, &voxel)) { const auto idx = static_cast<int>(voxel->dir); frequencies[idx]++; weights[idx] += voxel->dir_weight; }
  if (valid &= getVoxelAtPosition<TexVoxel>(gpu_hash_index_to_ptr, position - Vector3f(0.f, voxel_size, 0.f), block_size, &voxel)) { const auto idx = static_cast<int>(voxel->dir); frequencies[idx]++; weights[idx] += voxel->dir_weight; }
  if (valid &= getVoxelAtPosition<TexVoxel>(gpu_hash_index_to_ptr, position - Vector3f(0.f, 0.f, voxel_size), block_size, &voxel)) { const auto idx = static_cast<int>(voxel->dir); frequencies[idx]++; weights[idx] += voxel->dir_weight; }
  // clang-format on
  return valid;
}

template <typename BlockType>
__device__ inline int linearizedThreadVoxelIdx() {
  constexpr int voxels_per_block = BlockType::kVoxelsPerSide *
                                   BlockType::kVoxelsPerSide *
                                   BlockType::kVoxelsPerSide;
  constexpr int voxels_per_slice =
      BlockType::kVoxelsPerSide * BlockType::kVoxelsPerSide;
  // clang-format off
  return blockIdx.x * voxels_per_block 
          + threadIdx.z * voxels_per_slice 
          + threadIdx.y * BlockType::kVoxelsPerSide 
          + threadIdx.x;
  // clang-format on
}

__global__ void majorityVoteTexVoxelDirs(
    const Index3DDeviceHashMapType<TexBlock> gpu_hash_index_to_ptr,
    TexBlock** tex_block_ptrs, const Index3D* block_indices,
    const float block_size, const float voxel_size, TexVoxel::Dir* smooth_dirs,
    float* smooth_weights) {
  // Get the Voxels we'll check in this thread
  const TexBlock* tex_block = tex_block_ptrs[blockIdx.x];
  const Index3D& block_index = block_indices[blockIdx.x];
  Index3D voxel_index = Index3D(threadIdx.z, threadIdx.y, threadIdx.x);
  const TexVoxel* tex_voxel =
      &(tex_block->voxels[voxel_index[0]][voxel_index[1]][voxel_index[2]]);

  Vector3f position = getCenterPostionFromBlockIndexAndVoxelIndex(
      block_size, block_index, voxel_index);

  Vector7f weights = Vector7f::Zero();
  Vector7i frequencies = Vector7i::Zero();
  const bool valid_hist =
      computeDirHistogram(gpu_hash_index_to_ptr, position, block_size,
                          voxel_size, weights, frequencies);

  // write current dir to smoothed output
  const int linear_voxel_idx = linearizedThreadVoxelIdx<TexBlock>();
  smooth_dirs[linear_voxel_idx] = tex_voxel->dir;
  smooth_weights[linear_voxel_idx] = tex_voxel->dir_weight;

  if (!valid_hist) return;

  // find highest weight and second highest to determine if we overwrite the
  // TexVoxel direction.
  // TODO(rasaford) this is a very inefficient way to do an array partition.
  // Replace this with a better impelmentation
  int max_idx = -1, second_idx = -1;
  float max = -1.f, second = -1.f;
  for (int i = 0; i < 7; ++i) {
    if (weights[i] > max) {
      max_idx = i;
      max = weights[i];
    }
  }
  for (int i = 0; i < 7; ++i) {
    if (weights[i] < max && weights[i] > second) {
      second_idx = i;
      second = weights[i];
    }
  }

  // if the most common direction is twice as likely as the second most common
  // one, we update the current voxels direction to be the most common one
  constexpr float DIR_UPDATE_CONFIDENCE = .5f;
  if (DIR_UPDATE_CONFIDENCE * max > second && max > 0 && second > 0) {
    smooth_dirs[linear_voxel_idx] = static_cast<TexVoxel::Dir>(max_idx);
    smooth_weights[linear_voxel_idx] = max / frequencies[max_idx];
  }
}

__global__ void setTexVoxelDirs(const TexVoxel::Dir* dirs,
                                const float* smooth_weights,
                                TexBlock** tex_block_ptrs) {
  // Get the Voxel we'll check in this thread
  TexBlock* tex_block = tex_block_ptrs[blockIdx.x];
  Index3D voxel_index = Index3D(threadIdx.z, threadIdx.y, threadIdx.x);
  TexVoxel* tex_voxel =
      &(tex_block->voxels[voxel_index[0]][voxel_index[1]][voxel_index[2]]);

  const int linear_idx = linearizedThreadVoxelIdx<TexBlock>();
  const TexVoxel::Dir smooth_dir = dirs[linear_idx];
  if (tex_voxel->dir != smooth_dir) {
    tex_voxel->updateDir(smooth_dir, smooth_weights[linear_idx]);
  }
}

void updateTexVoxelDirectionsGPU(
    device_vector<const TsdfBlock*> neighbor_blocks,
    const device_vector<Index3D> block_indices,
    device_vector<TexBlock*>& tex_block_ptrs, const int num_blocks,
    const hipStream_t stream, const float block_size, const float voxel_size) {
  // Kernel call - One ThreadBlock launched per VoxelBlock
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const dim3 kThreadsPerBlock(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  const int num_thread_blocks = num_blocks;

  // Update TexVoxel directions in two steps:
  // 1. Update TexVoxel directions based on the TSDF surface gradient
  // 2. Comptue the smoothed directions with majority voting
  // 3. Update all TexVoxels to their smoothed directions
  //
  // We do this in two separate calls to guarantee that all TexVoxel directions
  // are set before smoothing.

  // clang-format off
  setTexVoxelDirsfromTsdfGradient<<<num_thread_blocks, kThreadsPerBlock, 0, stream>>>(
      neighbor_blocks.data(),
      tex_block_ptrs.data(),
      block_indices.data(),
      block_size,
      voxel_size
  );
  // clang-format on
  checkCudaErrors(hipStreamSynchronize(stream));
  checkCudaErrors(hipPeekAtLastError());

  // constexpr int voxels_per_block = TexBlock::kVoxelsPerSide *
  //                                  TexBlock::kVoxelsPerSide *
  //                                  TexBlock::kVoxelsPerSide;
  // device_vector<TexVoxel::Dir> smooth_dirs;
  // smooth_dirs.reserve(block_indices_device.size() * voxels_per_block);
  // device_vector<float> smooth_weights;
  // smooth_weights.reserve(block_indices_device.size() * voxels_per_block);

  // // clang-format off
  // majorityVoteTexVoxelDirs<<<num_thread_blocks, kThreadsPerBlock, 0,
  // stream>>>(
  //     tex_layer_view.getHash().impl_,
  //     tex_block_ptrs.data(),
  //     block_indices_device.data(),
  //     block_size,
  //     voxel_size,
  //     smooth_dirs.data(),
  //     smooth_weights.data()
  // );
  // // clang-format on
  // checkCudaErrors(hipStreamSynchronize(stream));
  // checkCudaErrors(hipPeekAtLastError());

  // // clang-format off
  // setTexVoxelDirs<<<num_thread_blocks, kThreadsPerBlock, 0, stream>>>(
  //   smooth_dirs.data(),
  //   smooth_weights.data(),
  //   tex_block_ptrs.data()
  // );
  // // clang-format on
  // checkCudaErrors(hipStreamSynchronize(stream));
  // checkCudaErrors(hipPeekAtLastError());
}

}  // namespace tex
}  // namespace nvblox